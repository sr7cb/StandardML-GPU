// #include "timer.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include <pthread.h>

// ==========================================================================
// context boilerplate


/* TODO: this stuff can probably go away entirely */

// struct futStuff {
//   struct futhark_context_config *cfg;
//   struct futhark_context *ctx;
// };

// void* futInit() {
//   struct timer_t t;
//   timer_begin(&t, "futInit");

//   struct futhark_context_config *cfg = futhark_context_config_new();
//   timer_report_tick(&t, "futhark_context_config_new");

//   struct futhark_context *ctx = futhark_context_new(cfg);
//   timer_report_tick(&t, "futhark_context_new");

//   struct futStuff *result = malloc(sizeof(struct futStuff));
//   result->cfg = cfg;
//   result->ctx = ctx;
//   return (void *)result;
// }

// void futFinish(struct futStuff * futStuff) {
//   struct futhark_context_config *cfg = futStuff->cfg;
//   struct futhark_context *ctx = futStuff->ctx;

//   struct timer_t t;
//   timer_begin(&t, "futFinish");

//   futhark_context_free(ctx);
//   timer_report_tick(&t, "futhark_context_free");

//   futhark_context_config_free(cfg);
//   timer_report_tick(&t, "futhark_context_config_free");

//   futStuff->ctx = NULL;
//   futStuff->cfg = NULL;
//   free(futStuff);
// }

// ==========================================================================
// dMM boilerplate


/* TODO: inputs and outputs for leaf DMM, dimension info, etc. */
struct dMMPackage {
  // struct futStuff *futStuff;  /* won't need this */

  /* need to be specialized for DMM */
  float * a;
  float * b;
  float * output;
  uint64_t inputLen;

  /* these should stay */
  bool finished;
  pthread_t friends;
};

/* TODO: call cublas */
void* asyncdMMFunc(void* rawArg) {
  // struct timer_t t;
  // timer_begin(&t, "asyncdMMFunc");

  struct dMMPackage *pack = (struct dMMPackage *)rawArg;

  // futhark_entry_add(pack->futStuff->ctx,
  //   &(pack->output),
  //   &(pack->outputLen), 
  //   pack->a, 
  //   pack->b);
  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);  
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, pack->inputLen, pack->inputLen, pack->inputLen, &alpha, (float*) pack->b, pack->inputLen, (float*) pack->a, pack->inputLen, &beta, (float*) pack->output, pack->inputLen);
  hipblasDestroy(handle);
  // futhark_context_sync(pack->futStuff->ctx);
  // timer_report_tick(&t, "done");
  pack->finished = true; /* VERY IMPORTANT! */
  return NULL;
}


/* TODO: build the package, but otherwise shouldn't need to change much. 
 *
 * (NOTE: futhark_new_... is essentially a memcpy, these need to be replaced
 *  with stuff for cublas)
 */
extern "C" struct dMMPackage * 
dMMSpawn(
  void * a,
  void * b,
  int64_t inputLen)
{
  // struct futhark_context *ctx = futStuff->ctx;
  struct dMMPackage *pack = (dMMPackage*)malloc(sizeof(struct dMMPackage));
  // pack->futStuff = futStuff;
  // pack->a = futhark_new_u8_1d(ctx, a, inputLen);
  hipMalloc(&(pack->a), inputLen*inputLen*sizeof(float));
  hipMemcpy(pack->a, (float*)a, inputLen*inputLen*sizeof(float), hipMemcpyHostToDevice);
  // pack->b = futhark_new_u8_1d(ctx, b, inputLen);
  hipMalloc(&(pack->b),  inputLen*inputLen*sizeof(float));
  hipMemcpy(pack->b, (float*)b,  inputLen*inputLen*sizeof(float), hipMemcpyHostToDevice);
  // pack->outputLen = 0;
  hipMalloc(&(pack->output),  inputLen*inputLen*sizeof(float));
  pack->inputLen = inputLen;
  pack->finished = false;

  if (0 != pthread_create(&(pack->friends), NULL, &asyncdMMFunc, pack)) {
    printf("ERROR: glue.c: futdMMSpawn: pthread_create failed\n");
    exit(1);
  }

  return pack;
}

/* TODO: probably doesn't need to change */
extern "C" uint8_t dMMPoll(struct dMMPackage *pack) {
  return pack->finished ? 1 : 0;
}

// int64_t futBigAddOutputSize(struct bigAddPackage *pack) {
//   // struct timer_t t;
//   // timer_begin(&t, "futPrimesOutputSize");

//   if (0 != pthread_join(pack->friend, NULL)) {
//     printf("ERROR: glue.c: futBigAddOutputSize: pthread_join failed\n");
//     exit(1);
//   }

//   // timer_report_tick(&t, "done");
//   return pack->outputLen;
// }

/* TODO: memcpy from GPU back to pack->output
 *
 * (NOTE: futhark_values is equivalent of this memcpy. needs to be replaced) */
extern "C" void dMMFinish(
  struct dMMPackage * pack,
  void * output)
{
  if (0 != pthread_join(pack->friends, NULL)) {
    printf("ERROR: glue.c: pthread_join failed\n");
    exit(1);
  }

  hipMemcpy(output, pack->output, pack->inputLen*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(pack->a);
  hipFree(pack->b);
  hipFree(pack->output);
  // futhark_values_u8_1d(pack->futStuff->ctx, pack->output, output);
  // futhark_free_u8_1d(pack->futStuff->ctx, pack->a);
  // futhark_free_u8_1d(pack->futStuff->ctx, pack->b);
  // futhark_free_u8_1d(pack->futStuff->ctx, pack->output);
  free(pack);

}
